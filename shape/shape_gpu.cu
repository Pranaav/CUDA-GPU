#include "hip/hip_runtime.h"
#include <cstddef>
#include <cstdlib>
#include <iostream>
#include "shape.hpp"


using std::cout;
using std::endl;
using std::vector;
using std::map;
using std::pair;

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)


void Shape::initForBFS(uint* &nov, uint* &adj)
{
    uint *h_nov, *h_adj;
    uint total = 0;
    const uint numv = triangles.size();
    for(auto i : adjacencyLists)
	{
		total += i.size();
	}
    const uint sizeV = (numv+1) * sizeof(uint),
                    sizeA = total * sizeof(uint);

    CUDA_CHECK_RETURN(hipHostMalloc(&h_nov, sizeV));
    CUDA_CHECK_RETURN(hipHostMalloc(&h_adj, sizeA));
    h_nov[0] = 0;

    uint k = 0;

    for(uint i = 0; i < numv; i++)
    {
        h_nov[i] = k;
        const uint size = adjacencyLists.at(i).size();
        for(uint j = 0; j < size; j++)
        {
            h_adj[k] = adjacencyLists.at(i).at(j);
            k++;
        }
    }
    h_nov[numv] = k;


    CUDA_CHECK_RETURN(hipMalloc(&nov, sizeV));
    CUDA_CHECK_RETURN(hipMalloc(&adj, sizeA));
    CUDA_CHECK_RETURN(hipMemcpy(nov, h_nov, sizeV, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(adj, h_adj, sizeA, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipHostFree(h_nov));
    CUDA_CHECK_RETURN(hipHostFree(h_adj));
}

bool Shape::findTriangle(float *p, uint &id)
{
	auto iter = pointmap.find(arrToPoint(p));
    if(iter == pointmap.end())
    {
        return false;

    }
	Triangle::sort(p);
    for(uint i : iter->second)
    {
        if(triangles.at(i)==p){
            id = i;
            return true;
        }
    }
    return false;
}
__global__
void _init1(bool *f, bool *s, bool *flag, const int u, const uint numv)
{
	uint i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < numv)
	{
		if (i == u)
		{
			f[i] = true;
			s[i] = true;
			*flag = false;
		}
		else
		{
			f[i] = false;
			s[i] = false;
		}
	}
}

__global__
void connected_kernel(const uint *nov, const uint *adj, const int dest, const int numv, bool *f, bool *s, bool *flag)
{
    uint i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numv && f[i])
    {
    	f[i] = false;
        uint v = nov[i];
        uint u = nov[i+1];
        for(uint j = v; j < u; j++)
        {
            int k = adj[j];
            if(!s[k])
            {
                f[k] = true;
                s[k] = true;
				flag[0] = false;
                if(k==dest)
                {
                	flag[1] = true;
                }
            }
        }
    }
}

bool Shape::isConnectedGpu(float *p, float *q)
{
    uint src,dest;
    if(!(findTriangle(p, src) && findTriangle(q, dest)))
    {
        return false;
    }
    if (src==dest) return true;
    uint *nov, *adj;
    initForBFS(nov, adj);

    bool *f, *s;
    bool *flag;
    const uint numv = triangles.size();
    uint size = numv * sizeof(bool);
    CUDA_CHECK_RETURN(hipMalloc(&f, size));
    CUDA_CHECK_RETURN(hipMalloc(&s, size));
    CUDA_CHECK_RETURN(hipMalloc(&flag, 2*sizeof(bool)));
    int tpb = 128;
    int bpg =(numv + tpb - 1) / tpb;

    _init1<<<bpg,tpb>>>(f, s, flag + 1, src, numv);
    CUDA_CHECK_RETURN(hipGetLastError());
    bool m[2]  = {true, false};
    while(true)
    {
    	CUDA_CHECK_RETURN(hipMemset(flag, true, sizeof(bool)));
        connected_kernel<<<bpg,tpb>>>(nov, adj, dest, numv, f, s, flag);
        CUDA_CHECK_RETURN(hipGetLastError());
        CUDA_CHECK_RETURN(hipMemcpy(m, flag, 2*sizeof(bool), hipMemcpyDeviceToHost));
        if(m[0] || m[1]) break;
    }

    CUDA_CHECK_RETURN(hipFree(flag));
    CUDA_CHECK_RETURN(hipFree(nov));
    CUDA_CHECK_RETURN(hipFree(adj));
    CUDA_CHECK_RETURN(hipFree(f));
    CUDA_CHECK_RETURN(hipFree(s));
    return m[1];
}

__global__
void _init2(bool *f, bool *nf, uint *col, const uint numv)
{
	uint i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < numv)
	{
		f[i] = true;
		nf[i] = false;
		col[i] = i;
	}
}

__global__
void colorcomponentsgpu(const uint* nov, const uint* adj, bool* f, bool* nf, uint* color,const uint numv, bool *flag)
{
	uint i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < numv)
	{
		if (f[i])
		{
			f[i] = false;
			uint hedcol = color[i];
			const uint ci = nov[i];
			const uint cj = nov[i + 1];
			bool ert = false;
			for (uint l = ci;l < cj;l++)
			{
				uint lefcol = color[adj[l]];
				if (lefcol > hedcol)
				{
					atomicMin(color + adj[l], hedcol);
					nf[adj[l]] = true;
					flag[0] = true;
				}
				else if (hedcol > lefcol)
				{
					ert = true;
					hedcol = lefcol;
				}
			}
			if (ert)
			{
				atomicMin(color + i, hedcol);
				nf[i] = true;
				flag[0] = true;
			}
		}
	}

}

__global__
void countcol(uint *col, uint* count, const uint numv)
{
	uint i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < numv)
	{
		if(col[i]==i)
		{
			atomicAdd((uint*)count, 1ull);
		}
	}
}

uint Shape::connectedComponentsGpu()
{
    uint *nov, *adj, *col;
    bool *f, *nf;
	bool m = true;
	bool *flag;
	const uint numv = triangles.size();
	const uint size = numv * sizeof(bool);
    initForBFS(nov, adj);
	CUDA_CHECK_RETURN(hipMalloc(&f, size));
	CUDA_CHECK_RETURN(hipMalloc(&nf, size));
	CUDA_CHECK_RETURN(hipMalloc(&col, numv * sizeof(uint)));
	CUDA_CHECK_RETURN(hipMalloc(&flag, sizeof(bool)));
	int tpb = 128;
	int bpg = (numv + tpb - 1) / tpb;
	_init2<<<bpg,tpb>>>(f,nf,col,numv);
	while(m)
	{
    	CUDA_CHECK_RETURN(hipMemset(flag, false, sizeof(bool)));
    	colorcomponentsgpu<<<bpg,tpb>>>(nov, adj, f, nf, col, numv, flag);
    	CUDA_CHECK_RETURN(hipGetLastError());
    	CUDA_CHECK_RETURN(hipMemcpy(&m, flag, sizeof(bool), hipMemcpyDeviceToHost));
    	bool *tmp = nf;
    	nf = f;
    	f = tmp;
	}
	uint ret;
	uint* count;
	CUDA_CHECK_RETURN(hipMalloc(&count, sizeof(uint)));
	countcol<<<bpg,tpb>>>(col, count, numv);
	CUDA_CHECK_RETURN(hipMemcpy(&ret, count, sizeof(uint), hipMemcpyDeviceToHost));


    CUDA_CHECK_RETURN(hipFree(flag));
    CUDA_CHECK_RETURN(hipFree(count));
    CUDA_CHECK_RETURN(hipFree(nov));
    CUDA_CHECK_RETURN(hipFree(adj));
    CUDA_CHECK_RETURN(hipFree(f));
    CUDA_CHECK_RETURN(hipFree(nf));
    CUDA_CHECK_RETURN(hipFree(col));
	return ret;
}

static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (-1);
}
